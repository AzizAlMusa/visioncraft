#include <hip/hip_runtime.h>

// CUDA kernel function to add two arrays
__global__ void addArrays(int* a, int* b, int* c, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        c[index] = a[index] + b[index];
    }
}

// Function to manage CUDA memory and run the kernel
void runAddArrays(int* a, int* b, int* c, int size) {
    int *d_a, *d_b, *d_c;

    // Allocate device memory
    hipMalloc((void **)&d_a, size * sizeof(int));  // size is the number of elements
    hipMalloc((void **)&d_b, size * sizeof(int));
    hipMalloc((void **)&d_c, size * sizeof(int));

    // Copy input arrays from host to device
    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel with 1 block and 'size' threads
    addArrays<<<1, size>>>(d_a, d_b, d_c, size);

    // Wait for the kernel to finish executing
    hipDeviceSynchronize();

    // Copy the result array from device to host
    hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // Free the device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
